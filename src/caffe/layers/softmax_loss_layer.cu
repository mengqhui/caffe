#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void SoftmaxWithLossLayer<Dtype, MItype, MOtype>::GenerateProgram() {
  this->device_program_ = this->device_->CreateProgram();
  stringstream ss;

  ss << this->device_program_->setup();
  ss << this->device_program_->template define_type<Dtype>("Dtype");
  ss << this->device_program_->template define_type<MItype>("MItype");
  ss << this->device_program_->template define_type<MOtype>("MOtype");
  ss << this->device_program_->template helper_functions<Dtype>();

  KernelArgs fw_args;
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "nthreads", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "prob_data", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "label", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "loss", KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "num", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "dim", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "spatial_dim", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<bool>(
                    "has_ignore_label", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "ignore_label", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "counts", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SoftmaxLossForwardGPU", fw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "index", "nthreads");
  ss << "const int_tp n = index / spatial_dim;" << std::endl;
  ss << "const int_tp s = index % spatial_dim;" << std::endl;
  ss << "const int_tp label_value = (int_tpc)(label[n * spatial_dim + s]);"
     << std::endl;
  ss << "if (has_ignore_label && label_value == ignore_label) {" << std::endl;
  ss << "loss[index] = (Dtype)0;" << std::endl;
  ss << "counts[index] = (Dtype)0;" << std::endl;
  ss << "} else {" << std::endl;
  ss << "loss[index] = -log("
     << "max((float)prob_data[n * dim + label_value * spatial_dim + s], "
     << "(float)FLT_MIN));" << std::endl;
  ss << "counts[index] = 1;" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  KernelArgs bw_args;
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "nthreads", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "top", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "label", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "bottom_diff", KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "num", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "dim", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "spatial_dim", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<bool>(
                    "has_ignore_label", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "ignore_label", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "counts", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SoftmaxLossBackwardGPU", bw_args);
  ss << "const int_tp channels = dim / spatial_dim;" << std::endl;
  ss << this->device_program_->kernel_loop("uint_tp", "index", "nthreads");
  ss << "const int_tp n = index / spatial_dim;" << std::endl;
  ss << "const int_tp s = index % spatial_dim;" << std::endl;
  ss << "const int_tp label_value = (int_tpc)(label[n * spatial_dim + s]);"
     << std::endl;
  ss << "if (has_ignore_label && label_value == ignore_label) {" << std::endl;
  ss << "for (int_tp c = 0; c < channels; ++c) {" << std::endl;
  ss << "bottom_diff[n * dim + c * spatial_dim + s] = 0;" << std::endl;
  ss << "}" << std::endl;
  ss << "counts[index] = 0;" << std::endl;
  ss << "} else {" << std::endl;
  ss << "bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;"
     << std::endl;
  ss << "counts[index] = 1;" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  this->device_program_->set_source(ss.str());
  this->device_program_->Compile(true, true);
}


template<typename Dtype, typename MItype, typename MOtype>
void SoftmaxWithLossLayer<Dtype, MItype, MOtype>::Forward_gpu(
    const vector<Blob<MItype>*>& bottom,
    const vector<Blob<MOtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);

  vptr<const Dtype> prob_data = prob_.gpu_data();
  vptr<const Dtype> label = bottom[1]->gpu_data();
  const int_tp dim = prob_.count() / outer_num_;
  const int_tp nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything, we use it here to avoid having
  // to allocate new GPU memory to accumulate intermediate results.
  vptr<Dtype> loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  vptr<Dtype> counts = prob_.mutable_gpu_diff();

  shared_ptr<DeviceKernel> kernel =
                      this->device_program_->GetKernel("SoftmaxLossForwardGPU");
  kernel->add_arg(&nthreads);
  kernel->add_arg(&prob_data);
  kernel->add_arg(&label);
  kernel->add_arg(&loss_data);
  kernel->add_arg(&outer_num_);
  kernel->add_arg(&dim);
  kernel->add_arg(&inner_num_);
  kernel->add_arg(&has_ignore_label_);
  kernel->add_arg(&ignore_label_);
  kernel->add_arg(&counts);

  vector<size_t> work_size(1, nthreads);
  vector<size_t> group;
  vector<size_t> local;
  this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
  kernel->Execute(group, local);

  Dtype loss;
  this->device_->template asum<Dtype>(nthreads, loss_data, &loss);
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID
      && has_ignore_label_) {
    this->device_->template asum<Dtype>(nthreads, counts, &valid_count);
  }
  top[0]->mutable_cpu_data()[0] = loss
      / get_normalizer(normalization_, valid_count);
  if (top.size() >= 2) {
    top[1]->ShareData(prob_);
  }

  // Clear scratch memory to prevent interfering with backward (see #6202).
  this->device_->template set<Dtype>(bottom[0]->count(), Dtype(0),
                                     bottom[0]->mutable_gpu_diff());
}

template<typename Dtype, typename MItype, typename MOtype>
void SoftmaxWithLossLayer<Dtype, MItype, MOtype>::Backward_gpu(
    const vector<Blob<MOtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<MItype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) <<
        this->type() << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    vptr<Dtype> bottom_diff = bottom[0]->mutable_gpu_diff();
    vptr<const Dtype> prob_data = prob_.gpu_data();
    vptr<const Dtype> top_data = top[0]->gpu_data();
    this->device_->template copy<Dtype>(prob_.count(), prob_data, bottom_diff);
    vptr<const Dtype> label = bottom[1]->gpu_data();
    const int_tp dim = prob_.count() / outer_num_;
    const int_tp nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    vptr<Dtype> counts = prob_.mutable_gpu_diff();

    shared_ptr<DeviceKernel> kernel =
                     this->device_program_->GetKernel("SoftmaxLossBackwardGPU");
    kernel->add_arg(&nthreads);
    kernel->add_arg(&top_data);
    kernel->add_arg(&label);
    kernel->add_arg(&bottom_diff);
    kernel->add_arg(&outer_num_);
    kernel->add_arg(&dim);
    kernel->add_arg(&inner_num_);
    kernel->add_arg(&has_ignore_label_);
    kernel->add_arg(&ignore_label_);
    kernel->add_arg(&counts);

    vector<size_t> work_size(1, nthreads);
    vector<size_t> group;
    vector<size_t> local;
    this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
    kernel->Execute(group, local);

    Dtype valid_count = -1;
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      this->device_->template asum<Dtype>(nthreads, counts, &valid_count);
    }
    const Dtype loss_weight = top[0]->cpu_diff()[0] /
    get_normalizer(normalization_, valid_count);
    this->device_->template scal<Dtype>(prob_.count(), loss_weight,
                                        bottom_diff);
  }
}

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, GenerateProgram,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, GenerateProgram,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, GenerateProgram,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Forward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Backward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SoftmaxWithLossLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
