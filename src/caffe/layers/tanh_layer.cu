#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/tanh_layer.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void TanHLayer<Dtype, MItype, MOtype>::GenerateProgram() {
  this->device_program_ = this->device_->CreateProgram();
  stringstream ss;

  ss << this->device_program_->setup();
  ss << this->device_program_->template define_type<Dtype>("Dtype");
  ss << this->device_program_->template define_type<MItype>("MItype");
  ss << this->device_program_->template define_type<MOtype>("MOtype");

  KernelArgs fw_args;
  fw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "n", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<MItype>(
                    "in", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<MOtype>(
                    "out", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("TanHForward", fw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "index", "n");
  ss << "out[index] = tanh(in[index]);" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  KernelArgs bw_args;
  bw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "n", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "in_diff", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "out_data", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "out_diff", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("TanHBackward", bw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "index", "n");
  ss << "Dtype tanhx = out_data[index];" << std::endl;
  ss << "out_diff[index] = in_diff[index] * (1 - tanhx * tanhx);"
         << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  this->device_program_->set_source(ss.str());
  this->device_program_->Compile(true, true);
}

template<typename Dtype, typename MItype, typename MOtype>
void TanHLayer<Dtype, MItype, MOtype>::Forward_gpu(
                                     const vector<Blob<MItype>*>& bottom,
                                     const vector<Blob<MOtype>*>& top) {
  vptr<const Dtype> bottom_data = bottom[0]->gpu_data();
  vptr<Dtype> top_data = top[0]->mutable_gpu_data();
  const int_tp count = bottom[0]->count();

  shared_ptr<DeviceKernel> kernel =
                                this->device_program_->GetKernel("TanHForward");
  kernel->add_arg(&count);
  kernel->add_arg(&bottom_data);
  kernel->add_arg(&top_data);

  vector<size_t> work_size(1, count);
  vector<size_t> group;
  vector<size_t> local;
  this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
  kernel->Execute(group, local);
}


template<typename Dtype, typename MItype, typename MOtype>
void TanHLayer<Dtype, MItype, MOtype>::Backward_gpu(
                                     const vector<Blob<MOtype>*>& top,
                                     const vector<bool>& propagate_down,
                                     const vector<Blob<MItype>*>& bottom) {
  if (propagate_down[0]) {
    vptr<const Dtype> top_data = top[0]->gpu_data();
    vptr<const Dtype> top_diff = top[0]->gpu_diff();
    vptr<Dtype> bottom_diff = bottom[0]->mutable_gpu_diff();
    const int_tp count = bottom[0]->count();

    shared_ptr<DeviceKernel> kernel =
                               this->device_program_->GetKernel("TanHBackward");
    kernel->add_arg(&count);
    kernel->add_arg(&top_diff);
    kernel->add_arg(&top_data);
    kernel->add_arg(&bottom_diff);

    vector<size_t> work_size(1, count);
    vector<size_t> group;
    vector<size_t> local;
    this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
    kernel->Execute(group, local);
  }
}

INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, GenerateProgram,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, GenerateProgram,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, GenerateProgram,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Forward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Backward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(TanHLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
