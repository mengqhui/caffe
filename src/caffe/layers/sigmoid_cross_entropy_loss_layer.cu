#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void SigmoidCrossEntropyLossLayer<Dtype, MItype, MOtype>::GenerateProgram() {
  this->device_program_ = this->device_->CreateProgram();
  stringstream ss;

  ss << this->device_program_->setup();
  ss << this->device_program_->template define_type<Dtype>("Dtype");
  ss << this->device_program_->template define_type<MItype>("MItype");
  ss << this->device_program_->template define_type<MOtype>("MOtype");

  KernelArgs fw_args;
  fw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "count", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "input_data", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "target", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<MItype>(
                    "loss", KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<bool>(
                    "has_ignore_label", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "ignore_label", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<MItype>(
                    "counts", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SigmoidCrossEntropyLossForwardGPU",
                                        fw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "i", "count");
  ss << "const int_tp target_value = (int_tpc)(target[i]);"
     << std::endl;
  ss << "if (has_ignore_label && target_value == ignore_label) {" << std::endl;
  ss << "loss[i] = (MItype)0;" << std::endl;
  ss << "counts[i] = (MItype)0;" << std::endl;
  ss << "} else {" << std::endl;
  ss << "loss[i] = input_data[i] * (target[i] -"
     << " ((input_data[i] >= (MItype)0) ? (MItype)1 : (MItype)0 )) -"
     << " (MItype)log((MItype)1 +"
     << " (MItype)exp(input_data[i] - (MItype)2 * input_data[i] *"
     << " ((input_data[i] >= (MItype)0) ? (MItype)1 : (MItype)0)));"
     << std::endl;
  ss << "counts[i] = (MItype)1;" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  KernelArgs bw_args;
  bw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "count", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<int_tp>(
                    "ignore_label", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "target", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "diff", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SigmoidCrossEntropyLossIgnoreDiffGPU",
                                        bw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "i", "count");
  ss << "const int_tp target_value = (int_tpc)(target[i]);"
     << std::endl;
  ss << "if (target_value == ignore_label) {" << std::endl;
  ss << "diff[i] = (MItype)0;" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  this->device_program_->set_source(ss.str());
  this->device_program_->Compile(true, true);
}

template<typename Dtype, typename MItype, typename MOtype>
void SigmoidCrossEntropyLossLayer<Dtype, MItype, MOtype>::Forward_gpu(
    const vector<Blob<MItype>*>& bottom,
    const vector<Blob<MOtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int_tp count = bottom[0]->count();
  const int_tp num = bottom[0]->num();
  // Stable version of loss computation from input data
  vptr<const MItype> input_data = bottom[0]->gpu_data();
  vptr<const MItype> target = bottom[1]->gpu_data();
  // Since this memory is not used for anything, we use it here to avoid having
  // to allocate new GPU memory to accumulate intermediate results.
  vptr<MItype> loss_data = bottom[0]->mutable_gpu_diff();

  MItype loss;
  vptr<MItype> count_data = bottom[1]->mutable_gpu_diff();
  MItype valid_count;

  shared_ptr<DeviceKernel> kernel =
          this->device_program_->GetKernel("SigmoidCrossEntropyLossForwardGPU");
  kernel->add_arg(&count);
  kernel->add_arg(&input_data);
  kernel->add_arg(&target);
  kernel->add_arg(&loss_data);
  kernel->add_arg(&has_ignore_label_);
  kernel->add_arg(&ignore_label_);
  kernel->add_arg(&count_data);

  vector<size_t> work_size(1, count);
  vector<size_t> group;
  vector<size_t> local;
  this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
  kernel->Execute(group, local);

  // Only launch another CUDA kernel if we actually need the valid count.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    this->device_->template asum<Dtype>(count, count_data, &valid_count);
  } else {
    valid_count = count;
  }
  this->device_->template asum<Dtype>(count, loss_data, &loss);
  normalizer_ = get_normalizer(normalization_, valid_count);
  top[0]->mutable_cpu_data()[0] = loss / normalizer_;

  // Clear scratch memory to prevent interfering with backward (see #6202).
  this->device_->template set<Dtype>(bottom[0]->count(),
                                     Dtype(0), bottom[0]->mutable_gpu_diff());
  this->device_->template set<Dtype>(bottom[1]->count(), Dtype(0),
                                     bottom[1]->mutable_gpu_diff());
}

template<typename Dtype, typename MItype, typename MOtype>
void SigmoidCrossEntropyLossLayer<Dtype, MItype, MOtype>::Backward_gpu(
    const vector<Blob<MOtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<MItype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL)<< this->type()
    << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    const int_tp count = bottom[0]->count();
    vptr<const Dtype> sigmoid_output_data = sigmoid_output_->gpu_data();
    vptr<const Dtype> target = bottom[1]->gpu_data();
    vptr<Dtype> bottom_diff = bottom[0]->mutable_gpu_diff();

    // First, compute the diff
    this->device_->template copy<Dtype>(count, sigmoid_output_data,
                                        bottom_diff);
    this->device_->template axpy<Dtype>(count, Dtype(-1), target, bottom_diff);
    // Zero out gradient of ignored targets.
    if (has_ignore_label_) {

      shared_ptr<DeviceKernel> kernel =
       this->device_program_->GetKernel("SigmoidCrossEntropyLossIgnoreDiffGPU");
      kernel->add_arg(&count);
      kernel->add_arg(&ignore_label_);
      kernel->add_arg(&target);
      kernel->add_arg(&bottom_diff);

      vector<size_t> work_size(1, count);
      vector<size_t> group;
      vector<size_t> local;
      this->device_->get_threads(&work_size, &group, &local, kernel.get(),
                                 true);
      kernel->Execute(group, local);
    }
    // Scale down gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer_;
    this->device_->template scal<Dtype>(count, loss_weight, bottom_diff);
  }
}

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, GenerateProgram,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, GenerateProgram,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, GenerateProgram,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Forward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Backward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidCrossEntropyLossLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
