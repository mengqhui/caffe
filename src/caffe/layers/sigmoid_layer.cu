#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/sigmoid_layer.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void SigmoidLayer<Dtype, MItype, MOtype>::GenerateProgram() {
  this->device_program_ = this->device_->CreateProgram();
  stringstream ss;

  ss << this->device_program_->setup();
  ss << this->device_program_->template define_type<Dtype>("Dtype");
  ss << this->device_program_->template define_type<MItype>("MItype");
  ss << this->device_program_->template define_type<MOtype>("MOtype");
  ss << this->device_program_->template helper_functions<Dtype>();

  KernelArgs fw_args;
  fw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "n", KERNEL_ARG_CONST));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "in", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  fw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "out", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SigmoidForward", fw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "index", "n");
  ss << "out[index] = (Dtype)0.5 * (Dtype)tanh((Dtype)0.5 * in[index])"
     << " + (Dtype)0.5;" << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  KernelArgs bw_args;
  bw_args.push_back(this->device_program_->template create_kernel_arg<uint_tp>(
                    "n", KERNEL_ARG_CONST));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "in_diff", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "out_data", KERNEL_ARG_CONST | KERNEL_ARG_GLOBAL_MEM));
  bw_args.push_back(this->device_program_->template create_kernel_arg<Dtype>(
                    "out_diff", KERNEL_ARG_GLOBAL_MEM));
  ss << this->device_program_->function("SigmoidBackward", bw_args);
  ss << this->device_program_->kernel_loop("uint_tp", "index", "n");
  ss << "const Dtype sigmoid_x = out_data[index];" << std::endl;
  ss << "out_diff[index] = in_diff[index] * sigmoid_x * ((Dtype)1 - sigmoid_x);"
     << std::endl;
  ss << "}" << std::endl;
  ss << "}" << std::endl;

  this->device_program_->set_source(ss.str());
  this->device_program_->Compile(true, true);
}

template<typename Dtype, typename MItype, typename MOtype>
void SigmoidLayer<Dtype, MItype, MOtype>::Forward_gpu(
                                        const vector<Blob<MItype>*>& bottom,
                                        const vector<Blob<MOtype>*>& top) {
  vptr<const Dtype> bottom_data = bottom[0]->gpu_data();
  vptr<Dtype> top_data = top[0]->mutable_gpu_data();
  const int_tp count = bottom[0]->count();

  shared_ptr<DeviceKernel> kernel =
                             this->device_program_->GetKernel("SigmoidForward");
  kernel->add_arg(&count);
  kernel->add_arg(&bottom_data);
  kernel->add_arg(&top_data);

  vector<size_t> work_size(1, count);
  vector<size_t> group;
  vector<size_t> local;
  this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
  kernel->Execute(group, local);
}


template<typename Dtype, typename MItype, typename MOtype>
void SigmoidLayer<Dtype, MItype, MOtype>::Backward_gpu(
                                        const vector<Blob<MOtype>*>& top,
                                        const vector<bool>& propagate_down,
                                        const vector<Blob<MItype>*>& bottom) {
  if (propagate_down[0]) {
    vptr<const Dtype> top_data = top[0]->gpu_data();
    vptr<const Dtype> top_diff = top[0]->gpu_diff();
    vptr<Dtype> bottom_diff = bottom[0]->mutable_gpu_diff();
    const int_tp count = bottom[0]->count();

    shared_ptr<DeviceKernel> kernel =
                            this->device_program_->GetKernel("SigmoidBackward");
    kernel->add_arg(&count);
    kernel->add_arg(&top_diff);
    kernel->add_arg(&top_data);
    kernel->add_arg(&bottom_diff);

    vector<size_t> work_size(1, count);
    vector<size_t> group;
    vector<size_t> local;
    this->device_->get_threads(&work_size, &group, &local, kernel.get(), true);
    kernel->Execute(group, local);
  }
}

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, GenerateProgram,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, GenerateProgram,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, GenerateProgram,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Forward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Backward_gpu,
                                  (half_fp), (half_fp), (half_fp));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(SigmoidLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
