
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_pooling_layer.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void CuDNNPoolingLayer<Dtype, MItype, MOtype>::Forward_gpu(
    const vector<Blob<MItype>*>& bottom,
    const vector<Blob<MOtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data().get_cuda_ptr();
  Dtype* top_data = top[0]->mutable_gpu_data().get_cuda_ptr();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template<typename Dtype, typename MItype, typename MOtype>
void CuDNNPoolingLayer<Dtype, MItype, MOtype>::Backward_gpu(
    const vector<Blob<MOtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<MItype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff().get_cuda_ptr();
  const Dtype* top_data = top[0]->gpu_data().get_cuda_ptr();
  const Dtype* bottom_data = bottom[0]->gpu_data().get_cuda_ptr();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff().get_cuda_ptr();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}


INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNPoolingLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNPoolingLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNPoolingLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNPoolingLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
#endif
