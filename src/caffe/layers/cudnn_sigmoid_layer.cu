
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_sigmoid_layer.hpp"

namespace caffe {

template<typename Dtype, typename MItype, typename MOtype>
void CuDNNSigmoidLayer<Dtype, MItype, MOtype>::Forward_gpu(const vector<Blob<MItype>*>& bottom,
    const vector<Blob<MOtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data().get_cuda_ptr();
  Dtype* top_data = top[0]->mutable_gpu_data().get_cuda_ptr();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
}

template<typename Dtype, typename MItype, typename MOtype>
void CuDNNSigmoidLayer<Dtype, MItype, MOtype>::Backward_gpu(
    const vector<Blob<MOtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<MItype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const Dtype* top_data = top[0]->gpu_data().get_cuda_ptr();
  const Dtype* top_diff = top[0]->gpu_diff().get_cuda_ptr();
  const Dtype* bottom_data = bottom[0]->gpu_data().get_cuda_ptr();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff().get_cuda_ptr();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
}


INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNSigmoidLayer, Forward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNSigmoidLayer, Forward_gpu,
                                  (double), (double), (double));

INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNSigmoidLayer, Backward_gpu,
                                  (float), (float), (float));
INSTANTIATE_CLASST_FUNC_3T_GUARDED(CuDNNSigmoidLayer, Backward_gpu,
                                  (double), (double), (double));

}  // namespace caffe
#endif
